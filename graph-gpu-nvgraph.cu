#include "hip/hip_runtime.h"
#include <stdio.h>
#include <nvgraph.h>
#include "common.h"
#include "utils.h"
#define ceiling(a,b) ( (a+b-1)/b )

const char* version_name = "nvGraph";
int NUM, id;
int N, M;
int T = 256;

typedef struct {
    nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphTraversalParameter_t traversal_param;
} nvGraph_t;

nvGraph_t* create_nvgraph(const dist_graph_t *graph, traverse_type_t traverse_type);

int *vis, *dis;
int *nowq, *nxtq;
int *v_pos, *e_dst;
int *pre, *in, *deg;
float *e_weight, *ans;
float *buf;

__global__ void preprocess_kernel( int *v_pos, int *e_dst, int *in, int N )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if( x < N ){
        int begin = v_pos[x];
        int end = v_pos[x+1];
        for(int e = begin; e < end; ++e) {
            int v = e_dst[e];
            atomicAdd( &in[v], 1 );
        }
    }
}

void preprocess(dist_graph_t *graph, traverse_type_t traverse_type) {
    if(graph->p_num > 1) {
        printf("not implemented. Only support single-process.\n");
        fatal_error(11);
    }
    graph->additional_info = create_nvgraph(graph, traverse_type);
    N = graph->global_v;
    M = graph->global_e;
    hipMalloc((void**)&v_pos, sizeof(int)*N );
    hipMalloc((void**)&e_dst, sizeof(int)*M );
    hipMalloc((void**)&e_weight, sizeof(float)*M );
    hipMalloc((void**)&vis, sizeof(int)*N );
    hipMalloc((void**)&dis, sizeof(int)*N );
    hipMalloc((void**)&ans, sizeof(float)*N );
    hipMalloc((void**)&nowq, sizeof(int)*N );
    hipMalloc((void**)&nxtq, sizeof(int)*N );
    hipMalloc((void**)&in, sizeof(int)*N );
    hipMalloc((void**)&deg, sizeof(int)*N );
    hipMalloc((void**)&buf, sizeof(float)*M );
    hipMalloc((void**)&pre, sizeof(int)*M );
    hipMemcpy( v_pos, graph->v_pos, sizeof(int)*N, hipMemcpyHostToDevice );
    hipMemcpy( e_dst, graph->e_dst, sizeof(int)*M, hipMemcpyHostToDevice );
    hipMemcpy( e_weight, graph->e_weight, sizeof(float)*M, hipMemcpyHostToDevice );

    dim3 grid_size (ceiling(N,T));
    dim3 block_size (T);
    preprocess_kernel<<<grid_size, block_size>>>(
        v_pos, e_dst, in, N
    );
    int *a = (int*)malloc( sizeof(int)*N );
    hipMemcpy( a, in, sizeof(int)*N, hipMemcpyDeviceToHost );
    int sum = 0;
    for( int i = 0; i < N; i ++ ){
        int tmp = sum;
        sum += a[i];
        a[i] = tmp;
    }
    hipMemcpy( in, a, sizeof(int)*N, hipMemcpyHostToDevice );
}

nvGraph_t* create_nvgraph(const dist_graph_t *graph, traverse_type_t traverse_type) {
    struct nvgraphCSRTopology32I_st CSR_input;
    hipDataType vertex_dimT[2] = {HIP_R_32I, HIP_R_32F};
    hipDataType edge_dimT[1] = {HIP_R_32F};
    nvGraph_t* nvgraph = (nvGraph_t*)malloc(sizeof(nvGraph_t));
    nvgraphCreate(&nvgraph->handle);
    nvgraphCreateGraphDescr(nvgraph->handle, &nvgraph->graph);
    CSR_input.nvertices = graph->global_v;
    CSR_input.nedges = graph->global_e;
    CSR_input.source_offsets = graph->v_pos;
    CSR_input.destination_indices = graph->e_dst;
    nvgraphSetGraphStructure(nvgraph->handle, nvgraph->graph, &CSR_input, NVGRAPH_CSR_32);
    if(traverse_type == BFS) {
        nvgraphAllocateVertexData(nvgraph->handle, nvgraph->graph, 1, vertex_dimT);
        nvgraphTraversalParameterInit(&nvgraph->traversal_param);
        nvgraphTraversalSetPredecessorsIndex(&nvgraph->traversal_param, 0);
        nvgraphTraversalSetUndirectedFlag(&nvgraph->traversal_param, true);
    } else {
        nvgraphAllocateVertexData(nvgraph->handle, nvgraph->graph, 2, vertex_dimT);
        nvgraphAllocateEdgeData  (nvgraph->handle, nvgraph->graph, 1, edge_dimT);
        nvgraphSetEdgeData(nvgraph->handle, nvgraph->graph, (void*)graph->e_weight, 0);
    }
    return nvgraph;
}

void destroy_additional_info(void *additional_info) {
    nvGraph_t* nvgraph = (nvGraph_t*)additional_info;
    nvgraphDestroyGraphDescr(nvgraph->handle, nvgraph->graph);
    nvgraphDestroy(nvgraph->handle);
    free(nvgraph);
    hipFree(v_pos);
    hipFree(e_dst);
}

__global__ void init_bfs( int *vis, int *dis, int N, int s )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if( x < N ){
        if( x == s ){
            vis[x] = x;
            dis[x] = 0;
        }
        else vis[x] = dis[x] = -1;
    }
}

__global__ void bfs_kernel( int *v_pos, int *e_dst, 
                             int *vis, int *dis, int cnt, 
                             int N, int *changed )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int flag = 0;
    __shared__ int fl;
    if( threadIdx.x == 0 ) fl = 0;
    //__syncthreads();
    if( x < N ){
        if( dis[x] == cnt ){
            int begin = v_pos[x];
            int end = v_pos[x+1];
            for(int e = begin; e < end; ++e) {
                int v = e_dst[e];
                if( vis[v] == -1 ){
                    vis[v] = x;
                    dis[v] = cnt + 1;
                    flag = 1;
                }
            }   
        } 
    }
    if( flag ){
        fl = 1;
    }
    __syncthreads();
    if( threadIdx.x == 0 && fl ) *changed = 1;
}

void bfs(dist_graph_t *graph, index_t s, index_t* pred) { 
    if(graph->p_id == 0){
        nvGraph_t* nvgraph = (nvGraph_t*)graph->additional_info;
        int *changed, cnt = 0, tmp = 0, tt = 0;
        hipMalloc((void **) &changed, sizeof(int));
        hipMemcpy( changed, &tt, sizeof(int), hipMemcpyHostToDevice );
        dim3 grid_size (ceiling(N,T));
        dim3 block_size (T);
        init_bfs<<<grid_size, block_size>>>( vis, dis, N, s );
        do{
            bfs_kernel<<<grid_size, block_size>>>(
                v_pos, e_dst, vis, dis, cnt, N, changed
            );
            cnt ++;
            hipMemcpy( &tmp, changed, sizeof(int), hipMemcpyDeviceToHost );
            //printf("ite %d %d\n", cnt, tmp);
            hipMemcpy( changed, &tt, sizeof(int), hipMemcpyHostToDevice );
        }while(tmp);

        hipMemcpy(pred, vis, sizeof(int)*N, hipMemcpyDeviceToHost);
        //printf("iteration %d %d root %d\n", cnt, s, pred[s]);
    }
}

__global__ void init_sssp( int *vis, float *ans, 
                           int *nowq, int *nxtq,
                           int N, int s )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if( x < N ){
        if( x == s ){
            vis[x] = x;
            ans[x] = 0.0;
            nowq[x] = 1;
        }
        else{
            vis[x] = -1;
            ans[x] = INFINITY;
            nowq[x] = 0;
        }
    }
}

__global__ void sssp_kernel( int *v_pos, int *e_dst, float *e_weight,
                             int *vis, float *ans,
                             int *nowq,
                             int *in, int *deg, float *buf, int *pre,
                             int cnt, int N, int *changed )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int flag = 0;
	int y = 0;
    __shared__ int fl;
    if( threadIdx.x == 0 ) fl = 0;
    //__syncthreads();
    if( x < N ){
        if( nowq[x] == cnt ){
            int begin = v_pos[x];
            int end = v_pos[x+1];
            for(int e = begin; e < end; ++e) {
                int v = e_dst[e];
				// int myturn = atomicAdd( &lock[v], 1 );
				// while( turn[v] != myturn ){
					// for( int i = 0; i < threadIdx.x; i ++ ) y ++;
				// }
				
				// while(1){
				// 	if( lock[v] == 1 ) continue;
                //     if( atomicCAS( &lock[v], 0, 1 ) == 0 ) break;
				// }
				// while( atomicCAS( &lock[v], 0, 1 ) != 0 ){
                //     //for( int i = 0; i < threadIdx.x; i ++ ) y ++;
                // }
				
				if( ans[v] > ans[x] + e_weight[e] ){

                    int pos = atomicAdd( &deg[v], 1 )+in[v];
                    buf[pos] = ans[x] + e_weight[e];
                    pre[pos] = x;

                    flag = 1;
                    
                }
				
                //lock[v] = 0;
				// atomicExch( &lock[v], 0 );
				//turn[v] ++;
            }   
        } 
    }
    if( flag ){
        fl = 1;
    }
    __syncthreads();
    if( threadIdx.x == 0 && fl ) *changed = 1;
}

__global__ void sssp_reduction( int *vis, float *ans,
                                int *in, int *deg, float *buf, 
                                int *pre, int *nowq,
                                int N, int cnt )
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    float min;
    int fa;
    min = INFINITY;
    fa = -1;
    if( x < N ){
        for( int i = in[x]; i < in[x]+deg[x]; i ++ ){
            if( buf[i] < min ){
                min = buf[i];
                fa = pre[i];
            }
        }
        if( deg[x] != 0 ){
            ans[x] = min;
            vis[x] = fa;
            nowq[x] = cnt+1;
        }
        deg[x] = 0;
    }
}

void sssp(dist_graph_t *graph, index_t s, index_t* pred, weight_t* distance){
    if(graph->p_id == 0){
        int *sp = (int*)malloc( sizeof(int)*N );
        float *fp = (float*)malloc( sizeof(float)*N );
        nvGraph_t* nvgraph = (nvGraph_t*)graph->additional_info;
        int *changed, cnt = 0, tmp = 0, tt = 0;
        hipMalloc((void **) &changed, sizeof(int));
        hipMemcpy( changed, &tt, sizeof(int), hipMemcpyHostToDevice );
        dim3 grid_size (ceiling(N,T));
        dim3 block_size (T);
        // for( int i = 0; i < M; i ++ ){
        //     if( graph->e_weight[i] <= 0.0 ) printf("%d %f\n", i, graph->e_weight[i] );
        // }
        init_sssp<<<grid_size, block_size>>>( vis, ans, nowq, nxtq, N, s );
        int num;
        do{
            cnt ++;
            //sssp_init_ans<<<grid_size, block_size>>>( deg, N );
            //printf("it %d", cnt);
            // hipMemcpy( sp, nowq, sizeof(int)*N, hipMemcpyDeviceToHost );
            // num = 0;
            // for( int i = 0; i < N; i ++ ){
            //     if( sp[i] == cnt ){
            //         //printf("%d ", i);
            //         num ++;
            //     }
            // }
            // printf(" num %d\n", num);
            
            // printf("ans\n");
            // num = 0;
            // hipMemcpy( fp, ans, sizeof(float)*N, hipMemcpyDeviceToHost );
            // for( int i = 0; i < N; i ++ ){
                // if( fp[i] != INFINITY ){
                    // printf("%d : %f\n", i, fp[i]);
                    // num ++;
                // }
            // }
            // printf("num %d\n", num);

            sssp_kernel<<<grid_size, block_size>>>(
                v_pos, e_dst, e_weight, vis, ans,
                nowq, in, deg, buf, pre, cnt, N, changed
            );
            //printf("nxtans\n");
            // num = 0;
            // hipMemcpy( fp, nxtans, sizeof(float)*N, hipMemcpyDeviceToHost );
            // for( int i = 0; i < N; i ++ ){
            //     if( fp[i] != INFINITY ){
            //         printf("%d : %f\n", i, fp[i]);
            //         num ++;
            //     }
            //     printf("%d %6d %.6f\n", cnt, i, fp[i]);
            // }
            // printf("num %d\n", num);

            // int debug[10];
            // hipMemcpy( debug, lock, sizeof(int)*2, hipMemcpyDeviceToHost );
            // printf("d0 %d d1 %d\n", debug[0], debug[1]);

            hipMemcpy( &tmp, changed, sizeof(int), hipMemcpyDeviceToHost );
            //printf("ite %d %d\n", cnt, tmp);
            hipMemcpy( changed, &tt, sizeof(int), hipMemcpyHostToDevice );

            sssp_reduction<<<grid_size, block_size>>>(
                vis, ans, in, deg, buf, pre, nowq, N, cnt
            );
            // int *tq = nowq;
            // nowq = nxtq;
            // nxtq = tq;
        }while(tmp);

        hipMemcpy(pred, vis, sizeof(int)*N, hipMemcpyDeviceToHost);
        hipMemcpy(distance, ans, sizeof(float)*N, hipMemcpyDeviceToHost);
        printf("iteration %d\n", cnt);
    }
}